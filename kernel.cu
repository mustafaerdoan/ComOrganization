#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define BLOCK_SIZE 16
#define ARRAY_SIZE 1280

int main(int argc, char const *argv[])
{
	// allocate in host
	int *h_a, *h_b, *h_c1, *h_c2;
	hipHostMalloc((void **)&h_a, sizeof(int)*ARRAY_SIZE*ARRAY_SIZE);
	hipHostMalloc((void **)&h_b, sizeof(int)*ARRAY_SIZE*ARRAY_SIZE);
	hipHostMalloc((void **)&h_c1, sizeof(int)*ARRAY_SIZE*ARRAY_SIZE);
	hipHostMalloc((void **)&h_c2, sizeof(int)*ARRAY_SIZE*ARRAY_SIZE);

	for (int i = 0; i < ARRAY_SIZE; ++i) {
		for (int j = 0; j < ARRAY_SIZE; ++j) {
			h_a[i * ARRAY_SIZE + j] = rand() % 100;
			h_b[i * ARRAY_SIZE + j] = rand() % 100;
		}
	}
	
	// Allocate in device 
	int *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a, sizeof(int)*ARRAY_SIZE*ARRAY_SIZE);
	hipMalloc((void **)&d_b, sizeof(int)*ARRAY_SIZE*ARRAY_SIZE);
	hipMalloc((void **)&d_c, sizeof(int)*ARRAY_SIZE*ARRAY_SIZE);

	// copy from host to device memory
	hipMemcpy(d_a, h_a, sizeof(int)*ARRAY_SIZE*ARRAY_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(int)*ARRAY_SIZE*ARRAY_SIZE, hipMemcpyHostToDevice);

	int grid_rows = BLOCK_SIZE*BLOCK_SIZE;
	int grid_cols = ARRAY_SIZE / grid_rows;

	dim3 dimGrid(grid_cols, grid_cols,1);
	dim3 dimBlock(grid_rows, grid_rows,1);

	float elapsed_time_gpu;

	//description to calculate time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// start time of GPU
	hipEventRecord(start, 0);

	matrix_mult_gpu <<<dimGrid, dimBlock >> > (d_a, d_b, d_c, ARRAY_SIZE);
	
	// copy from device to host 
	hipMemcpy(h_c1, d_c, sizeof(int)*ARRAY_SIZE*ARRAY_SIZE, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	// stop time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// compute time elapse on GPU computing
	hipEventElapsedTime(&elapsed_time_gpu, start, stop);
	printf("Time elapsed on matrix multiplician %d on GPU: %.1f s.\n\n", ARRAY_SIZE, elapsed_time_gpu);

	// start the CPU version
	clock_t t;
	t = clock();
	matrix_mult_cpu(h_a, h_b, h_c2, ARRAY_SIZE);
	t = clock() - t;
	double elapsed_time = ((double)t) / CLOCKS_PER_SEC;
	printf("Time elapsed on matrix multiplication %d on CPU: %.1f s.\n\n", ARRAY_SIZE, elapsed_time);

	
	
	// free memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c1);
	hipHostFree(h_c2);
	return 0;
}


__global__ void matrix_mult_gpu(int *a, int *b, int *c, int n)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int sum = 0;
	if (col < n && row < n)
	{
		for (int i = 0; i < n; i++)
		{
			sum += a[row * n + i] * b[i * n + col];
		}
		c[row * n + col] = sum;
	}
}

void matrix_mult_cpu(int *h_a, int *h_b, int *h_result, int n) 
{
	int i, j, k;
	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
		{
			h_result[i*n + j] = 0;
			for (k = 0; k < n; k++)
			{
				h_result[i*n + j] += h_a[k + i * n] * h_b[k*n + j];
			}
		}
	}
}